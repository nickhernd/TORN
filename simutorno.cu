#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "simutorno.h"
#include <chrono>

// Variables globales
int PuntosVueltaHelicoide;
float PasoHelicoide;

// Función para crear la superficie
void CrearSuperficie(TSurf* S, int u, int v) {
    S->UPoints = u;
    S->VPoints = v;
    
    // Reservar memoria para el array 2D
    S->Buffer = (TPoints3D**)malloc(v * sizeof(TPoints3D*));
    for (int i = 0; i < v; i++) {
        S->Buffer[i] = (TPoints3D*)malloc(u * sizeof(TPoints3D));
    }
}

// Función para liberar la memoria de la superficie
void BorrarSuperficie(TSurf* S) {
    if (S->Buffer != NULL) {
        for (int i = 0; i < S->VPoints; i++) {
            free(S->Buffer[i]);
        }
        free(S->Buffer);
        S->Buffer = NULL;
    }
    S->UPoints = 0;
    S->VPoints = 0;
}

// Función para leer la superficie desde un archivo
int LeerSuperficie(char* filename, TSurf* S) {
    FILE* f = fopen(filename, "r");
    if (f == NULL) {
        printf("Error al abrir el archivo %s\n", filename);
        return 0;
    }
    
    // Variables para leer datos
    char line[256];
    int sectionNum = 0, pointsPerSection = 0;
    float step = 0;
    int pointsPerRound = 0;
    
    // Leer parámetros
    while (fgets(line, sizeof(line), f)) {
        if (strstr(line, "SECTION NUMBER") != NULL) {
            sscanf(line, "%*[^:]: %d", &sectionNum);
        } else if (strstr(line, "POINTS PER SECTION") != NULL) {
            sscanf(line, "%*[^:]: %d", &pointsPerSection);
        } else if (strstr(line, "STEP") != NULL) {
            sscanf(line, "%*[^:]: %f", &step);
        } else if (strstr(line, "POINTS PER ROUND") != NULL) {
            sscanf(line, "%*[^:]: %d", &pointsPerRound);
        } else if (strstr(line, "POINTS") != NULL) {
            // Empiezan los datos de puntos
            break;
        }
    }
    
    // Guardar variables globales
    PasoHelicoide = step;
    PuntosVueltaHelicoide = pointsPerRound;
    
    // Crear superficie
    CrearSuperficie(S, sectionNum, pointsPerSection);
    
    // Leer puntos
    for (int v = 0; v < S->VPoints; v++) {
        for (int u = 0; u < S->UPoints; u++) {
            float x, y, z;
            if (fscanf(f, "%f %f %f", &x, &y, &z) != 3) {
                printf("Error en formato de archivo\n");
                fclose(f);
                return 0;
            }
            S->Buffer[v][u].x = x;
            S->Buffer[v][u].y = y;
            S->Buffer[v][u].z = z;
        }
    }
    
    fclose(f);
    return 1;
}

// Implementación en CPU de la simulación
void SimulacionTornoCPU(TSurf S, float** CPUBufferMenorY) {
    // Reserva de la malla de salida (mismos puntos que la original)
    *CPUBufferMenorY = (float*)malloc(S.UPoints * S.VPoints * sizeof(float));

    // Rotación total (360 grados en radianes)
    //float anguloTotal = 2.0f * M_PI;
    int pasos = PuntosVueltaHelicoide;

    // Para cada punto de la superficie
    for (int v = 0; v < S.VPoints; v++) {
        for (int u = 0; u < S.UPoints; u++) {
            float y_original = S.Buffer[v][u].y;
            float z_original = S.Buffer[v][u].z;

            float minY = y_original;

            // Simular la rotación sobre el eje X
            for (int i = 1; i <= pasos; i++) {
                float angulo = i * PasoHelicoide;  // PasoHelicoide ya viene en radianes

                // Rotación sobre el eje X: solo cambian Y y Z
                float y_rot = y_original * cos(angulo) - z_original * sin(angulo);
                float z_rot = y_original * sin(angulo) + z_original * cos(angulo);

                if (y_rot < minY) {
                    minY = y_rot;
                }
            }

            // Guardar el valor mínimo de Y encontrado para este punto
            (*CPUBufferMenorY)[v * S.UPoints + u] = minY;
        }
    }
}

// Kernel CUDA para la simulación
__global__ void tornoKernel(TPoints3D* buffer, float* menorY, 
                           int uPoints, int vPoints, 
                           int puntosVuelta, float paso) {

    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if (u >= uPoints || v >= vPoints) return;

    int idx = v * uPoints + u;

    float y_original = buffer[idx].y;
    float z_original = buffer[idx].z;
    float minY = y_original;

    for (int i = 1; i <= puntosVuelta; i++) {
        float angulo = i * paso;

        float y_rot = y_original * cosf(angulo) - z_original * sinf(angulo);

        if (y_rot < minY) {
            minY = y_rot;
        }
    }

    menorY[idx] = minY;
    
}

// Implementación en GPU de la simulación
void SimulacionTornoGPU(TSurf S, float** GPUBufferMenorY) {
    int numPuntos = S.UPoints * S.VPoints;

    // Reservar memoria en GPU
    TPoints3D* GPU_buffer;
    float* GPU_menorY;
    // Asignamos memoria en la GPU para la malla de puntos (S.Buffer)
    hipMalloc((void**)&GPU_buffer, numPuntos * sizeof(TPoints3D));
    // Asignamos memoria en la GPU para el buffer de resultados
    hipMalloc((void**)&GPU_menorY, numPuntos * sizeof(float));

    // Crear array plano de TPoints3D para copiar desde la CPU
    TPoints3D* buffer_plano = (TPoints3D*)malloc(numPuntos * sizeof(TPoints3D));
    for (int v = 0; v < S.VPoints; ++v) {
        for (int u = 0; u < S.UPoints; ++u) {
            buffer_plano[v * S.UPoints + u] = S.Buffer[v][u];
        }
    }

    // Copiar superficie a GPU
    hipMemcpy(GPU_buffer, buffer_plano, numPuntos * sizeof(TPoints3D), hipMemcpyHostToDevice);

    // Definir bloques e hilos
    dim3 blockSize(16, 16);  //16x16 es un estandar razonable, pero se podría buscar otras configuraciónes según el tamaño de la malla
    dim3 gridSize((S.UPoints + blockSize.x - 1) / blockSize.x,
                  (S.VPoints + blockSize.y - 1) / blockSize.y);

    // Ejecutar kernel de simulación
    tornoKernel<<<gridSize, blockSize>>>(GPU_buffer, GPU_menorY,
                                         S.UPoints, S.VPoints,
                                         PuntosVueltaHelicoide, PasoHelicoide);

    // Esperar finalización y chequear errores
    hipDeviceSynchronize();

    // Reservar salida en CPU
    *GPUBufferMenorY = (float*)malloc(numPuntos * sizeof(float));
    // Copiar resultados a CPU
    hipMemcpy(*GPUBufferMenorY, GPU_menorY, numPuntos * sizeof(float), hipMemcpyDeviceToHost);

    // Liberar recursos
    free(buffer_plano);
    hipFree(GPU_buffer);
    hipFree(GPU_menorY);
}

// Función de prueba
void runTest(char* filename) {
    TSurf superficie;
    float *CPUBuffer = NULL, *GPUBuffer = NULL;

    if (!LeerSuperficie(filename, &superficie)) {
        printf("Error leyendo la superficie\n");
        return;
    }
    
    // Ejecutar CPU
    //TODO: Registrar tiempo ejecución CPU    
    auto start_cpu = std::chrono::high_resolution_clock::now();
    SimulacionTornoCPU(superficie, &CPUBuffer);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration_CPU = end_cpu - start_cpu;

    // Ejecutar GPU
    //TODO: Registrar tiempo ejecución GPU
    auto start_gpu = std::chrono::high_resolution_clock::now();
    SimulacionTornoGPU(superficie, &GPUBuffer);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration_GPU = end_gpu - start_gpu;

    // Comparar resultados
    int errores = 0;
    int total = superficie.UPoints * superficie.VPoints;
    for (int i = 0; i < total; i++) {
        if (fabs(CPUBuffer[i] - GPUBuffer[i]) > 1e-3) {
            errores++;
            if (errores < 10)  // mostrar los primeros errores
                printf("Error en punto %d: CPU = %f, GPU = %f\n", i, CPUBuffer[i], GPUBuffer[i]);
        }
		/*else 
			printf("Correcto punto %d: CPU = %f, GPU = %f\n", i, CPUBuffer[i], GPUBuffer[i]);*/
    }

    if (errores == 0){
        //printf("¡Correcto! CPU y GPU coinciden.\n");
        printf("Tiempo en CPU | Tiempo en GPU\n");
        printf("-----------------------------\n");        
        printf("    %f  | %f \n", duration_CPU, duration_GPU);
    }
    else
        printf("Diferencias detectadas en %d puntos\n", errores);

    free(CPUBuffer);
    free(GPUBuffer);
    BorrarSuperficie(&superficie);
}

// Programa principal
int main(int argc, char** argv) {
    if (argc < 2) {
        printf("Uso: %s archivo.for\n", argv[0]);
        return 1;
    }
    
    runTest(argv[1]);
    return 0;
}
